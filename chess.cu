#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
	
#define BOXES 64
#define PIECES 16
#define SIDE 8

__device__ int black[BOXES][PIECES + 1];
__device__ int white[BOXES][PIECES + 1];
__device__ char piece[PIECES + 1][BOXES + 1];
__device__ int piece_pos[PIECES+1];
__device__ volatile unsigned int numBlocks = 0;
__device__ volatile unsigned int numBlocks2 = 0;

__device__ void printBlack(){
	
	printf("black\n");

		for(int i = 0; i < BOXES; i++){
			printf("i: %d ",i);
			for(int j = 0; j <= PIECES; j++){
				if(black[i][j] == 1)
				printf("%d",black[i][j]);
				else
				printf("_");
			}
			printf("\n");
		}
}

__device__ void printWhite(){
		
		printf("white\n");
		
		for(int i = 0; i < BOXES; i++){
			printf("i: %d ",i);
			for(int j = 0; j <= PIECES; j++){
				if(white[i][j] == 1)
				printf("%d",white[i][j]);
				else
				printf("_");
			}
			printf("\n");
		}
}

__device__ void updateDS(int curr_piece, int x, int y, int val){
	
	int row = x*SIDE + y;
	int col = (curr_piece % 100);
	
	if(curr_piece < 100){
		//fill white DS
		white[row][col] = val;				

	} else {
		//fill black DS
		black[row][col] = val;

	}
		
}

__global__ void canAnyoneCome(int row){
	int col = threadIdx.x + 1;
	
	if(threadIdx.x == 0){
		black[row][0] = 0;
		white[row][0] = 0;
	}
	
	if(black[row][col] == 1){
		black[row][0] = 1;
	}
	
	if(white[row][col] == 1){
		white[row][0] = 1;
	}
	
	
}

__global__ void king(int *board, int pos, int turn){
	int id = threadIdx.x;
	if(id < 8 && board[pos] != 0){
		int i = pos/SIDE;
		int j = pos%SIDE;
		int curr_piece = board[pos];
		
		int x=0,y=0;
		int a,b;
		//setting the value of new position
		if(id == 0 || id == 1 || id == 2){
			
			a = -1;
			if(id == 0)
				b = -1;
			else if(id == 2)
				b = 0;
			else
				b = 1;
		
		}else if( id == 3 || id == 4 || id == 5){
			
			a = 1;
			if(id == 3)
				b = -1;
			else if(id == 4)
				b = 0;
			else
				b = 1;
			
		}else{
			a = 0;
			if(id == 6)
				b = -1;
			else
				b = 1;
		
		}
		
		 
		x = i + a;
		y = j + b;
	
		if((x < SIDE && x >= 0) && (y < SIDE && y >= 0)){
		
			updateDS(curr_piece,x,y,0);
		
			int new_piece = board[x*SIDE + y];
			int is_valid = 0;
	
			if(new_piece == 0){
				//place blank -> valid move
				is_valid = 1;
			}else{
				if((curr_piece < 100 && new_piece > 100) || (curr_piece > 100 && new_piece < 100)){
					//opposition here -> valid move
					is_valid = 1;
				}else if(turn == 0 && new_piece < 100 && curr_piece < 100){
					is_valid = 1;
				}else if(turn == 1 && new_piece > 100 && curr_piece > 100){
					is_valid = 1;
				}
			}
			
			if(is_valid)
				updateDS(curr_piece,x,y,1);

		}
	}
}


__global__ void queen(int *board, int pos, int turn){
	int id = threadIdx.x;
	
	if(id < 8 && board[pos] != 0){
		int i = pos/SIDE;
		int j = pos%SIDE;
		int curr_piece = board[pos];
		
		int x=0,y=0;
		int a,b;
		//setting the value of new position
		if(id == 0 || id == 1 || id == 2){
			
			a = -1;
			if(id == 0)
				b = -1;
			else if(id == 2)
				b = 0;
			else
				b = 1;
		
		}else if( id == 3 || id == 4 || id == 5){
			
			a = 1;
			if(id == 3)
				b = -1;
			else if(id == 4)
				b = 0;
			else
				b = 1;
			
		}else{
			a = 0;
			if(id == 6)
				b = -1;
			else
				b = 1;
		
		}
		
		 
		x = i + a;
		y = j + b;
		int flag = 0;	
		
		while((x < SIDE && x >= 0) && (y < SIDE && y >= 0) ){
			
			updateDS(curr_piece,x,y,0);
			
			int new_piece = board[x*SIDE + y];
			int is_valid = 0;
			
			if(new_piece == 0){
				//place blank -> valid move
				is_valid = 1;
			}else{
				if((curr_piece < 100 && new_piece > 100) || (curr_piece > 100 && new_piece < 100)){
					//opposition here -> valid move
					is_valid = 1;
				}else if(turn == 0 && new_piece < 100 && curr_piece < 100){
					is_valid = 1;
				}else if(turn == 1 && new_piece > 100 && curr_piece > 100){
					is_valid = 1;
				}
				flag = 1;
			}
			
			if(is_valid)
				updateDS(curr_piece,x,y,1);
				
			if(flag)
				break;
			else{
				x += a;
				y += b;
			}
		}	
	}
}


__global__ void knight(int *board, int pos, int turn){
	int id = threadIdx.x;
	
	if(id < 8 && board[pos] != 0){
		int i = pos/SIDE;
		int j = pos%SIDE;
		int curr_piece = board[pos];
		
		int x=0,y=0;
		//setting the value of new position
		if(id == 0 || id == 1){
			
			x = i - 2;
			if(id == 0)
				y = j - 1;
			else
				y = j + 1;
		
		}else if( id == 2 || id == 3){
			
			x = i + 2;
			if(id == 2)
				y = j - 1;
			else
				y = j + 1;
			
		}else if( id == 4 || id == 5){
			
			y = j - 2;
			if(id == 4)
				x = i - 1;
			else
				x = i + 1;
			
		}else{
			
			y = j + 2;
			if(id == 6)
				x = i - 1;
			else
				x = i + 1;
		
		}
		
		if((x < SIDE && x >= 0) && (y < SIDE && y >= 0)){
			
			updateDS(curr_piece,x,y,0);
			
			int new_piece = board[x*SIDE + y];
			int is_valid = 0;
			
			if(new_piece == 0){
				//place blank -> valid move
				is_valid = 1;
			}else{
				if((curr_piece < 100 && new_piece > 100) || (curr_piece > 100 && new_piece < 100)){
					//opposition here -> valid move
					is_valid = 1;
				}else if(turn == 0 && new_piece < 100 && curr_piece < 100){
					is_valid = 1;
				}else if(turn == 1 && new_piece > 100 && curr_piece > 100){
					is_valid = 1;
				}
			}
			
			if(is_valid)
				updateDS(curr_piece,x,y,1);
		}
	}
	
}


__global__ void bishop(int *board,int pos, int turn){
	int id = threadIdx.x;
	if( id < 4 && board[pos]!=0 ){
		int i = pos/SIDE;
		int j = pos%SIDE;
		int curr_piece = board[pos];
		
		int x=0,y=0;
		int a,b;
		
		//Assigning Initial Values of position
		if( id == 0 ){
			//For north east pos
			a = -1;
			b = 1;
			
		}else if( id == 1){
			//For north west pos
			a = -1;
			b = -1;
		
		}else if( id == 2){
			//For south east pos 
			a = 1;
			b = 1;
		
		}else if( id == 3){
			//For south west pos
			a = 1;
			b = -1;
		
		}
		x = i + a;
		y = j + b;
		int flag = 0;
		while( (x < SIDE && x >= 0) && (y < SIDE && y >= 0) ){
			
			//printf("thread:%d %d %d\n",id, x, y);
			
			updateDS(curr_piece,x,y,0);
			
			int is_valid = 0;
			int new_piece = board[x*SIDE + y];
			
			if( new_piece == 0){
				//Valid Move
				is_valid = 1;
				
			} else {
				if( (new_piece < 100 && curr_piece > 100) || (new_piece < 100 && curr_piece > 100) ){
					//Valid Move and STOPPING the loop
					is_valid = 1;				
				}else if(turn == 0 && new_piece < 100 && curr_piece < 100){
					is_valid = 1;
				}else if(turn == 1 && new_piece > 100 && curr_piece > 100){
					is_valid = 1;
				}
				flag = 1;
			
			}
			
			if( is_valid )
				updateDS(curr_piece,x,y,1);
				
			if( flag == 1 )
					break;
					
			x+=a;
			y+=b;
		}
	}
}


__global__ void rook(int* board,int pos,int turn){
	int id = threadIdx.x;
	if( id < 4 && board[pos]!=0 )
	{
		int i = pos/SIDE;
		int j = pos%SIDE;
		int curr_piece = board[pos];
		
		int x,y;
		int itr= 1;
		int flag = 0;
		
		//Assigning initial values to x and y
		if( id == 0 ){
			//For left pos
			x = i;
			y = j - itr;
		}else if( id == 1){
			//For right pos
			x = i;
			y = j + itr;
		}else if( id == 2){
			//For upper pos 
			x = i - itr;
			y = j;
		}else if( id == 3){
			//For lower pos
			x = i + itr;
			y = j;
		}
		itr = itr + 1;
		while((x < SIDE && x >= 0) && (y < SIDE && y >= 0) ){
			
			updateDS(curr_piece,x,y,0);
			
			int is_valid = 0;		
			int new_piece = board[x*SIDE + y];
			
			if( new_piece == 0){
				//Valid Move
				is_valid = 1;
			}
			else{
				if( (new_piece < 100 && curr_piece > 100) || (new_piece < 100 && curr_piece > 100) )
				{
					//Valid Move and STOPPING the loop
					is_valid = 1;				
				}else if(turn == 0 && new_piece < 100 && curr_piece < 100){
					is_valid = 1;
				}else if(turn == 1 && new_piece > 100 && curr_piece > 100){
					is_valid = 1;
				}
				flag = 1;
			}
			
			if( is_valid )
				updateDS(curr_piece,x,y,1);
				
			if( flag == 1 )
					break;
			
			//Finding the new position in x and y
			if( id == 0 )
			{
				//For left pos
				x = i;
				y = j - itr;
			}else if( id == 1){
				//For right pos
				x = i;
				y = j + itr;
			}else if( id == 2){
				//For upper pos 
				x = i - itr;
				y = j;
			}else if( id == 3){
				//For lower pos
				x = i + itr;
				y = j;
			}
			itr++;

		}
	}
}
	

__global__ void pawn(int *board,int pos,int turn){
	
	int id  = threadIdx.x;
	
	int i = pos/SIDE;
	int j = pos%SIDE;

	int curr_piece = board[pos];
	
	//x=row y=col of new position 
	
	int x,y;
	
	if(curr_piece > 100){
		x = i+1;
		y = j+(id-1);
		
	}else{
		x = i-1;
		y = j+(id-1);
		
	}
	
	if((x < SIDE && x >= 0) && (y < SIDE && y >= 0)){
		
		updateDS(curr_piece,x,y,0);
		
		int new_piece = board[x*SIDE + y];
		int is_valid = 0;
		if( j == y ){
			//blank in front
			if(new_piece == 0){
				
				//valid move
				if((turn == 0 && curr_piece > 100) || (turn == 1 && curr_piece < 100))
					is_valid = 1;
				
			}
		} else {
			//piece off opposite side
			if(	(new_piece > 100 && curr_piece < 100)||  
				(new_piece < 100 && new_piece != 0 && curr_piece > 100)){
				//valid move
				is_valid = 1;		
			}else if((turn == 0 && curr_piece < 100) ||
					 (turn == 1 && curr_piece > 100)){  //turn of black but piece is white
				is_valid = 1;
			}
			
			
		}
		
		if(is_valid){
			updateDS(curr_piece,x,y,1);
		}
	}  
	
}


__global__ void markUnsafe(int col, int turn){
			
		int row = threadIdx.x;
		
		if(turn == 0){
			if(black[col][row + 1] == 1){
				piece[row+1][col+1] = 'U';
				piece[row+1][0] = 'p';	//moves present
			}
		}else{
			if(white[col][row + 1] == 1){
				piece[row+1][col+1] = 'U';
				piece[row+1][0] = 'p';	//moves present
			}
		}
		
		
}

__global__ void markSafe(int col, int turn){
			
		int row = threadIdx.x;
		
		if(turn == 0){
			if(black[col][row + 1] == 1){
				piece[row+1][col+1] = 'S';
				piece[row+1][0] = 'p';	//moves present
			}
		}else{
			if(white[col][row + 1] == 1){
				piece[row+1][col+1] = 'S';
				piece[row+1][0] = 'p';	//moves present
			}
		}
		
		
}

__global__ void markAttackingUnsafe(int col, int turn){
			
		int row = threadIdx.x;
		
		if(turn == 0){
			if(black[col][row + 1] == 1){
				piece[row+1][col+1] = 'A';
				piece[row+1][0] = 'p';	//moves present
			}
		}else{
			if(white[col][row + 1] == 1){
				piece[row+1][col+1] = 'A';
				piece[row+1][0] = 'p';	//moves present
			}
		}
		
}

__global__ void markAttackingSafe(int col, int turn){
			
		int row = threadIdx.x;
		
		if(turn == 0){
			if(black[col][row + 1] == 1){
				piece[row+1][col+1] = 'X';
				piece[row+1][0] = 'p';	//moves present
			}
		}else{
			if(white[col][row + 1] == 1){
				piece[row+1][col+1] = 'X';
				piece[row+1][0] = 'p';	//moves present
			}
		}
		
		
}


__device__ void markCurrentUnsafe(int *board, int curr_pos){
	
	int row = board[curr_pos] % 100;
	int col = curr_pos + 1;
	
	piece[row][col] = '?';
	
}


__device__ void markCurrentSafe(int *board, int curr_pos){
	int row = board[curr_pos] % 100;
	int col = curr_pos + 1;
	
	piece[row][col] = '#';
}

__global__ void computeMoves(int *board,int turn){
	//pos of a piece
	
	int curr_pos = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	if(curr_pos < BOXES && board[curr_pos] != 0){
		
		//val of a piece
		int curr_piece = board[curr_pos];
		
		//num corresponding piece
		int piece = curr_piece % 100;
		
		switch(piece){
			
			case 1:
			case 2:
			case 3:
			case 4:
			case 5:
			case 6:
			case 7:
			case 8:
				pawn<<<1,3>>>(board,curr_pos,turn);
				break;
			
			case 9:
			case 10:
				rook<<<1,4>>>(board,curr_pos,turn);
				break;
			
			case 11:
			case 12:
				knight<<<1,8>>>(board,curr_pos,turn);
				break;
			
			case 13:
			case 14:
				bishop<<<1,4>>>(board,curr_pos,turn);
				break;
			
			case 15:
				queen<<<1,8>>>(board,curr_pos,turn);
				break;
				
			case 16: 
				king<<<1,8>>>(board,curr_pos,turn);
				break;
			
		}
		
	}
	
	__syncthreads();
	//synchronize across all blocks
	
	if(threadIdx.x == 0){
		atomicAdd((unsigned int*)&numBlocks, 1);
	}
	
	while(numBlocks != gridDim.x);
	
	
	/*for all now select the safe moves
	 *
	 *  S=> safe
	 * 	U=> unsafe
	 * 	X=> attacking safe
	 * 	A=> attacking unsafe
	 * 
	 * 	turn
	 * 		0=> black
	 * 		1=> white 
	 * 
	 */ 
	
	
	//which positions black and white can come  
	
	canAnyoneCome<<<1,16>>>(curr_pos);
	if(threadIdx.x == 0){
		atomicAdd((unsigned int *)&numBlocks2, 1);
		numBlocks = 0;
	}
	
	while(numBlocks2 != gridDim.x);
	
	int col = curr_pos;
	
	hipDeviceSynchronize();
	
	//black turn
	if(turn == 0){
		
		if(board[curr_pos] == 0){
			
			//unsafe if white can come
			if(white[curr_pos][0] == 1){
				//unsafe
				markUnsafe<<<1,16>>>(col,turn);
				
			} else {
				//safe
				markSafe<<<1,16>>>(col,turn);
			}
		}else{
		
			//black can come and not zero means white is there
			//attacking
			if(black[curr_pos][0] == 1){
				
				//if other white can come there
				if(white[curr_pos][0] == 1){
					//unsafe
					markAttackingUnsafe<<<1,16>>>(col,turn);
				} else {
					//safe
					markAttackingSafe<<<1,16>>>(col,turn);
				}

			} else if(board[curr_pos] > 100){ //black is present there
				//if other white can come there
				if(white[curr_pos][0] == 1){
					//unsafe
					markCurrentUnsafe(board,curr_pos);
				} else {
					//safe
					markCurrentSafe(board,curr_pos);
				}
			}
		}
	} else {
		
		if(board[curr_pos] == 0){
			
			//unsafe if black can come
			if(black[curr_pos][0] == 1){
				//unsafe
				markUnsafe<<<1,16>>>(col,turn);
			} else {
				//safe
				markSafe<<<1,16>>>(col,turn);
			}
			
		}else{
			
			//white can come and not zero means black is there
			//attacking
			if(white[curr_pos][0] == 1){
				
				//if other black can come there
				if(black[curr_pos][0] == 1){
					//unsafe
					markAttackingUnsafe<<<1,16>>>(col,turn);
				} else {
					//safe
					markAttackingSafe<<<1,16>>>(col,turn);
				}
			
			} else if(board[curr_pos] < 100){
				
				//if other black can come there
				if(black[curr_pos][0] == 1){
					//unsafe
					markCurrentUnsafe(board,curr_pos);
				} else {
					//safe
					markCurrentSafe(board,curr_pos);
				}
			
			}
			
		}
	}
	
	__syncthreads();
	if(threadIdx.x == 0){
		atomicAdd((unsigned int *)&numBlocks, 1);
		numBlocks2 = 0;
	}
	
	while(numBlocks != gridDim.x);
	
	
	/*
	if(threadIdx.x == 0)
	for(int i=0; i <= PIECES; i++){
		for(int j=0; j <= BOXES; j++){
			printf("%d\t", piece[i][j]);
		}
		printf("\n");
	}
	
	*/
	
	if(turn == 0){
		if(board[curr_pos] > 100){
			int piece = board[curr_pos] %100;
			piece_pos[piece] = curr_pos;
		}
	} else {
		if(board[curr_pos] < 100 && board[curr_pos] != 0){
			int piece = board[curr_pos] %100;
			piece_pos[piece] = curr_pos;
		}
	}
	
	
	__syncthreads();
	if(threadIdx.x == 0){
		atomicAdd((unsigned int *)&numBlocks2, 1);
		numBlocks = 0;
	}
	
	while(numBlocks2 != gridDim.x);
	hipDeviceSynchronize();
	/*
	if(threadIdx.x == 0)
	for(int i=0; i <= PIECES; i++){
		printf("i:%d ", i);
		for(int j=0; j <= BOXES; j++){
			printf("%c-", piece[i][j]);
		}
		printf("\n");
	}
	
	if(threadIdx.x == 0)
	printf("\n");
	
	if(threadIdx.x == 0)	
	for(int i=0; i <= PIECES; i++){
	printf("%d ", piece_pos[i]);
	}
	
	*/
	
	
	if(threadIdx.x  == 0 && blockIdx.x ==0){
		printf("\n");
		printf("# => represents your piece Currently Safe \n");
		printf("? => represents your piece Currently UnSafe \n");
	
		printf("S => represents a Safe Move \n");
		printf("U => represents an Unsafe Move \n");
		printf("A => represents Attacking Unsafe Move \n");
		printf("X => represents Attacking Safe Move \n");
		printf("\n");
		
		for(int i = 0; i < 16 ; i++){
			int pos = piece_pos[i+1];
			int dice = board[pos]%100;
			int val = dice;
			//int x = piece_pos[dice];
			
			if(turn == 0)
				val = dice + 100;
			
			//printf("%d , %d , %c\n",dice, val,piece[dice][0]);
			
			if(piece[dice][0] == 'p'){
				
				printf("possible moves for piece: %d\n", val);
				
				for(int i=0; i < SIDE; i++){
					for(int j=0; j < SIDE; j++){
						
						if(piece[dice][ SIDE*i + j + 1] == '\0'){
							printf("- ");
						}else
							printf("%c ", piece[dice][ SIDE*i + j + 1]);
					}
					printf("\n");
				}
				
				printf("\n");
				
			}
		}
		
	}
	
	
	//chk for all attacking places
	
	/*
	if(curr_pos == 0){
		printBlack();
		printWhite();
	}
	*/
}

/*
 * white piece 1-16
 * 			1-8 	pawns
 * 			9-10 	rook
 * 			11-12 	knight
 * 			13-14	bishop
 * 			15		Queen
 * 			16		King
 * 
 * Black Side	101 - 116 (correponding values)
 */

int main(void){
	
	
	int *h_board,*d_board,turn;
	
	/* 	turn
	 * 		0=> black
	 * 		1=> white 
	 * 
	 */ 
	
	
	scanf("%d",&turn);
	
	h_board = (int *) malloc(BOXES * sizeof(int));
	
	for(int i=0; i < SIDE; i++){
		for(int j=0; j < SIDE; j++){
			scanf("%d", &h_board[ SIDE*i + j]);
		}
	}	
	
	hipMalloc(&d_board, BOXES * sizeof(int));
	
	
	hipMemcpy(d_board, h_board, BOXES * sizeof(int) , hipMemcpyHostToDevice);
	
	computeMoves<<<8,8>>>(d_board,turn);
	
	hipDeviceSynchronize();
	
	
	printf("Current Board Position \n\n");
	
	for(int i=0; i < SIDE; i++){
		for(int j=0; j < SIDE; j++){
			//printf("%d\t", h_board[ SIDE*i + j]);
			
			int piece = h_board[ SIDE*i + j] % 100;
			
			switch(piece){
			
			case 1:
			case 2:
			case 3:
			case 4:
			case 5:
			case 6:
			case 7:
			case 8:
				if(h_board[ SIDE*i + j] < 100)
					printf("p ");
				else
					printf("P ");
				break;
			
			case 9:
			case 10:
				if(h_board[ SIDE*i + j] < 100)
					printf("r ");
				else
					printf("R ");
				break;
			
			case 11:
			case 12:
				if(h_board[ SIDE*i + j] < 100)
					printf("h ");
				else
					printf("H ");
				break;
			
			case 13:
			case 14:
				if(h_board[ SIDE*i + j] < 100)
					printf("b ");
				else
					printf("B ");
				break;
			
			case 15:
				if(h_board[ SIDE*i + j] < 100)
					printf("q ");
				else
					printf("Q ");
				break;
				
			case 16: 
				if(h_board[ SIDE*i + j] < 100)
					printf("k ");
				else
					printf("K ");
				break;
			default:
				printf("- ");
				break;
			}
			
		}
		printf("\n");
	}
	
	return 0;
}
